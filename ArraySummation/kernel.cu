#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


__global__ void reduction_neighbored_pairs(int * input, int * temp, int size)
{

}

int main()
{

	printf("RUNNING neighborded pairs reduction kernel\n");

	int size = 1 << 27;
	int byte_size = size * sizeof(int);

	int block_size = 128;

	int* h_input, * href;
	h_input = (int*)malloc(byte_size);
	initialize(h_input, size, INIT_RANDOM);

	int cpu_result = reduction_cpu(h_input, size);

	dim3 block(block_size);
	dim3 grid(size / block.x);

	int temp_array_byte_size = sizeof(int) * grid.x;

	h_ref = (int*)malloc(temp_array_byte_size);

	int* d_input, * d_temp;

	hipMalloc((void**)&d_input, byte_size);


	hipDeviceReset();
	return 0;
}
